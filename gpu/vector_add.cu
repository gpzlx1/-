#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>


#define N   100000

__global__ void add(int *a, int *b, int *c) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += step;
	}
}

int main(void) {
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;

	a = (int*)malloc(N * sizeof(int));
	b = (int*)malloc(N * sizeof(int));
	c = (int*)malloc(N * sizeof(int));

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	//init
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	hipMemcpy(dev_a, a, N * sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int),
		hipMemcpyHostToDevice);
	
	time_t begin, end;
	begin = clock();
	{
		add << <512, 512 >> > (dev_a, dev_b, dev_c);
		hipDeviceSynchronize();
	}
	end = clock();

	printf("GPU time = %d ms\n", int(end - begin));

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	bool success = true;
	for (int i = 0; i < N; i++) {
		if ((a[i] + b[i]) != c[i]) {
			printf("Error:  %d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}

	if (success)    
		printf("Everything is ok!\n");

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	free(a);
	free(b);
	free(c);

	return 0;
}


